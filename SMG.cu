#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <limits>
//#include "ReadRadar.h"

#include "opencv2/core/core.hpp"
#include <opencv2/imgproc/imgproc.hpp>
#include "opencv2/highgui/highgui.hpp"
#include <sl/Camera.hpp>

#include<sys/resource.h>
#include <time.h>
#include<sys/time.h>

#include "Comparison.h"
#include "Kernels.cuh"

using namespace std;
using namespace cv;

#define threadx 16
#define thready 16

#define USECAMARA false
#define Camara 1
#define WIDTHIMAGE 1280
#define LENGTHIMAGE 720

__host__ void SMG();
__host__ void AggregateCostCom(int* cost, int* L, int width, int length,int maxDisparity, int directionx, int directiony);
__host__ int minBetweenNumbersInt(int a, int b, int c, int d);
__host__ Mat DisparitySelectionP(int* L2, int* L4, int* L5, int* L7,int* L1, int* L3, int* L6, int* L8, int maxDisparity, int width, int length);
__host__ void DivideImagesCam(Mat completeImage, Mat* left, Mat* right);

__host__ pointCoo getPoint(int ID, int pathX, int pathY, int width, int length);
__host__ void getKernelInitialInformation(int pathNumber, startInfo* pixelDi, int width, int length);

//sl::Camera zed;

int main (int argc, char** argv){
	//Initialising Radar
	//ReadRadar radar;
	//radar.Connect();
	//radar.Read();
	//Initialising Radar


	//Comparison Calculations
	//Comparison comp;
	//comp.CompareDisparities();
	//------------------------

	SMG();
	return 0;
}

__host__ void SMG(){
	struct timeval timstr;
	Mat left, right;
	Mat completeImage;
	//Texture Creation
	Mat leftD,rightD;
	uchar* imageLeftA,*imageRightA,*leftC,*rightC;
	startInfo* initialInfo;

	//CensusDeclarations
	unsigned int* censusLa;
	unsigned int* censusRa;
	int* costK;
	/*int* L1;
	int* L2;
	int* L3;
	int* L4;
	int* L5;
	int* L6;
	int* L7;
	int* L8;*/

	//Maximum box value is depending on bytes used
	int BoxCostX = 9;
	int BoxCostY = 7;
	int maxDisparity = 100;


#if USECAMARA
	VideoCapture stream(Camara);
	for (int i =0; i<100;i++){
		stream.read(completeImage);
	}

	DivideImagesCam(completeImage,&right,&left);
#else
	//Reading Images
	left = imread("Images/KITTY/left/0000000000.png", CV_LOAD_IMAGE_COLOR);
	right= imread("Images/KITTY/right/0000000000.png", CV_LOAD_IMAGE_COLOR);

#endif

	//Converting Images
	Mat leftBlack;
	cvtColor( left, leftBlack, CV_BGR2GRAY );
	Mat rightBlack;
	cvtColor( right, rightBlack, CV_BGR2GRAY );

	//Passing images to kernels----------------------------
	imageLeftA = (uchar*)malloc((sizeof(uchar))*(left.cols)*(left.rows));
	imageRightA= (uchar*)malloc((sizeof(uchar))*(right.cols)*(right.rows));
	imageLeftA = leftBlack.data;
	imageRightA= rightBlack.data;
	hipMalloc(&leftC,(sizeof(uchar))*(left.cols)*(left.rows));
	hipMalloc(&rightC,(sizeof(uchar))*(left.cols)*(left.rows));
	hipMemcpy(leftC,imageLeftA,(sizeof(uchar))*(left.cols)*(left.rows),hipMemcpyHostToDevice);
	hipMemcpy(rightC,imageRightA,(sizeof(uchar))*(left.cols)*(left.rows),hipMemcpyHostToDevice);
	//Done passing images to kernels------------------------

	int decreseX = BoxCostX/2 + BoxCostX/2;
	int decreseY = BoxCostY/2 + BoxCostY/2;

	int dimX = ((leftBlack.cols-decreseX) / threadx);
	int dimY = ((leftBlack.rows-decreseY) / thready);
	dim3 dimGrid(dimX,dimY);
	dim3 dimBlock(threadx,thready);

	//Timing
	gettimeofday(&timstr, NULL);
	double begin = timstr.tv_sec + (timstr.tv_usec / 1000000.0);

	//First Kernel, Census and cost Computation
	hipMalloc(&censusLa,(sizeof(unsigned int))*(left.cols-decreseX)*(left.rows-decreseY));
	hipMalloc(&censusRa,(sizeof(unsigned int))*(right.cols-decreseX)*(right.rows-decreseY));
	hipMalloc(&costK,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));

	KernelDisparityCalculations<<<dimGrid,dimBlock>>>(BoxCostX,BoxCostY,censusLa,censusRa,left.cols-decreseX,left.rows-decreseY,leftC,rightC,costK);
	hipDeviceSynchronize();

	//------------Done-First-Kernel-----------------------------


	//Second Kernel, Semi global matching and disparity Selection
	int widthR = left.cols-decreseX;
	int lengthR = left.rows-decreseY;
	int threadNum = 10;
	int pathNumber =((widthR+lengthR-1)*4 + widthR *2 + lengthR*2);
	dim3 dimGrid2(pathNumber);
	dim3 dimBlock2(threadNum);
	//Assigning Paths
	/*hipMalloc(&L1,((sizeof(int))*(widthR)*(lengthR)*(maxDisparity+1)));
	hipMalloc(&L2,((sizeof(int))*(widthR)*(lengthR)*(maxDisparity+1)));
	hipMalloc(&L3,((sizeof(int))*(widthR)*(lengthR)*(maxDisparity+1)));
	hipMalloc(&L4,((sizeof(int))*(widthR)*(lengthR)*(maxDisparity+1)));
	hipMalloc(&L5,((sizeof(int))*(widthR)*(lengthR)*(maxDisparity+1)));
	hipMalloc(&L6,((sizeof(int))*(widthR)*(lengthR)*(maxDisparity+1)));
	hipMalloc(&L7,((sizeof(int))*(widthR)*(lengthR)*(maxDisparity+1)));
	hipMalloc(&L8,((sizeof(int))*(widthR)*(lengthR)*(maxDisparity+1)));
	*/
	initialInfo = (startInfo*)malloc((sizeof(startInfo))*pathNumber);
	getKernelInitialInformation(pathNumber,initialInfo,widthR,lengthR);
	KernelSemiGlobal<<<dimGrid2,dimBlock2>>>(costK, widthR, lengthR);
	hipDeviceSynchronize();
	//------------Done-Second-Kernel-----------------------------

	int* cost = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	hipMemcpy(cost,costK,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1),hipMemcpyDeviceToHost);
	//Cost From GPU to host
	cout << "Done CUDA " << endl;

	//Aggregate Cost
	int* L1 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L1,left.cols-decreseX,left.rows-decreseY,maxDisparity,-1,-1);
	cout << "Done" << endl;

	int* L2 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L2,left.cols-decreseX,left.rows-decreseY,maxDisparity,0,-1);
	cout << "Done" << endl;

	int* L3 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L3,left.cols-decreseX,left.rows-decreseY,maxDisparity,1,-1);
	cout << "Done" << endl;

	int* L4 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L4,left.cols-decreseX,left.rows-decreseY,maxDisparity,-1,0);
	cout << "Done" << endl;

	int* L5 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L5,left.cols-decreseX,left.rows-decreseY,maxDisparity,1,0);
	cout << "Done" << endl;

	int* L6 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L6,left.cols-decreseX,left.rows-decreseY,maxDisparity,-1,1);
	cout << "Done" << endl;

	int* L7 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L7,left.cols-decreseX,left.rows-decreseY,maxDisparity,0,1);
	cout << "Done" << endl;

	int* L8 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L8,left.cols-decreseX,left.rows-decreseY,maxDisparity,1,1);
	cout << "Done" << endl;




	//Disparity Selection
	Mat disparity = DisparitySelectionP(L1,L2,L3,L4,L5,L6,L7,L8,maxDisparity,left.cols-decreseX,left.rows-decreseY);
	gettimeofday(&timstr, NULL);
	double end = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
	printf("Elapsed time Local Matching:\t\t\t%.6lf (s)\n", end - begin);


	namedWindow("SMG");
	imshow("SMG",disparity);
	//imwrite("disparity.png",disparity);
	waitKey(0);


	//free Memory
	hipFree(leftC);
	hipFree(rightC);
	hipFree(censusLa);
	hipFree(censusRa);
	hipFree(costK);
	free(cost);
	free(imageLeftA);
	free(imageRightA);
	free(L1);
	free(L2);
	free(L3);
	free(L4);
	free(L5);
	free(L6);
	free(L7);
	free(L8);
	free(initialInfo);
}

__host__ pointCoo getPoint(int ID, int pathX, int pathY, int width, int length){
	pointCoo point;
	int startX;
	int startY;
	//Initial Point, this is the corner of the diagonals
	if(ID == 0){
		startX = 0;
		startY = 0;
		if(pathX == -1){
			startX = width - 1;
		}
		if(pathY == -1){
			startY = length - 1;
		}
	}
	//As diagonals go through the width and length, this needs to be divided into two segments
	//Segment 1 across the x axis of the image, fix y coordinate of the image
	else if (ID < width){
		startX = ID;
		startY = 0;
		if(pathX == -1){
			startX = startX - 1;
		}
		if(pathY == -1){
			startY = length-1;
		}
	}
	//Segment 2 across the y axis of the image, fix x coordinate
	else if (ID < width + length - 1){
		int newID = ID - width;
		startX = 0;
		startY = newID;
		if (pathX == -1){
			startX = width -1;
		}
		if (pathY == -1){
			startY = startY -1;
		}
	}

	point.x = startX;
	point.y = startY;
	return point;
}

__host__ void getKernelInitialInformation(int pathNumber, startInfo* pixelDi, int width, int length) {
	for (int blockID = 0; blockID < pathNumber; blockID++) {
		int ID = blockID;
		int LA = blockID; // Location in Array

		if (blockID < width) {
			pixelDi[LA].startX = ID;
			pixelDi[LA].startY = 0;
			pixelDi[LA].directionX = 0;
			pixelDi[LA].directionY = 1;
		} else if (blockID < 2 * width) {
			ID = blockID - width;
			pixelDi[LA].startX = ID;
			pixelDi[LA].startY = length - 1;
			pixelDi[LA].directionX = 0;
			pixelDi[LA].directionY = -1;
		} else if (blockID < 2 * width + length) {
			ID = blockID - 2 * width;
			pixelDi[LA].startX = 0;
			pixelDi[LA].startY = ID;
			pixelDi[LA].directionX = 1;
			pixelDi[LA].directionY = 0;
		} else if (blockID < 2 * width + 2 * length) {
			ID = blockID - 2 * width - length;
			pixelDi[LA].startX = width - 1;
			pixelDi[LA].startY = ID;
			pixelDi[LA].directionX = -1;
			pixelDi[LA].directionY = 0;
		} else if (blockID < 2 * width + 2 * length + (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length;
			pixelDi[LA].directionX = 1;
			pixelDi[LA].directionY = 1;
			pointCoo point = getPoint(ID, 1, 1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else if (blockID < 2 * width + 2 * length + 2 * (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length - (width + length - 1);
			pixelDi[LA].directionX = -1;
			pixelDi[LA].directionY = -1;
			pointCoo point = getPoint(ID, -1, -1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else if (blockID < 2 * width + 2 * length + 3 * (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length - 2 * (width + length - 1);
			pixelDi[LA].directionX = 1;
			pixelDi[LA].directionY = -1;
			pointCoo point = getPoint(ID, 1, -1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else if (blockID < 2 * width + 2 * length + 4 * (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length - 3 * (width + length - 1);
			pixelDi[LA].directionX = -1;
			pixelDi[LA].directionY = 1;
			pointCoo point = getPoint(ID, -1, 1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else {
			printf("ifs are wrong \n");
		}
	}
}


//Disparity Selection Process
__host__ Mat DisparitySelectionP(int* L2, int* L4, int* L5, int* L7,int* L1, int* L3, int* L6, int* L8, int maxDisparity, int width, int length){
	Mat disparity(length,width,CV_8U);

	for(int y=0; y<length; y++){
		for(int x=0; x<width; x++){
			int costA = 99999;
			int disPix = 0;
			for (int d = 0; d<maxDisparity; d++){
				int sumAgg = L1[width*(y+d*length)+x] + L2[width*(y+d*(length))+x]+
						L3[width*(y+d*length)+x] + L4[width*(y+d*(length))+x] + L5[width*(y+d*(length))+x]+
						L6[width*(y+d*length)+x] + L7[width*(y+d*(length))+x] + L8[width*(y+d*length)+x];
				if (sumAgg < costA){
					costA = sumAgg;
					disPix = d;
				}
 			}
			//cout << "disparity pixel " <<x << " " << y << " is " << disPix << " with value "<< costA<< endl;
			disparity.at<uchar>(y,x) = disPix;
		}
	}
	return disparity;
}
//Gets the cost Computation Across all Paths
__host__ void AggregateCostCom(int* cost, int* L, int width, int length, int maxDisparity, int directionx, int directiony){
	//penalties
	int p1 = 10;
	int p2 = 100;

	int startX,startY, increaseX, increaseY;

	if(directionx <= 0){
		startX = 0;
		increaseX = 1;
	}
	else{
		startX = width-1;
		increaseX = -1;
	}
	if(directiony <= 0){
		startY = 0;
		increaseY = 1;
	}
	else{
		startY = length-1;
		increaseY = -1;
	}

	int* minimuns = (int*)malloc((sizeof(int))*(width)*(length));
	int x;
	int y = startY;

	for(int yC=0; yC<length; yC++){
		int influenceY = y + directiony;
		x = startX;
		for(int xC=0; xC<width; xC++){
			int influenceX = x + directionx;

			int minimunValue = 999999;

			if(influenceX > width || influenceX < 0 || influenceY > length || influenceY < 0){
				for (int d = 0; d<maxDisparity; d++){
					int costPixel = cost[width*(y+d*(length))+x];
					L[width*(y+d*(length))+x] = costPixel;
					if(costPixel < minimunValue){
						minimunValue = costPixel;
						//cout << "minimum ->  " << costPixel;
					}
				}
			}
			else{
				/*cout << x << " " << y <<endl;
				waitKey(0);*/
				for (int d = 0;  d<maxDisparity; d++){
					int costPixel = cost[width*(y+d*(length))+x];
					int currentD = L[width*(influenceY+d*(length))+influenceX];
					int previousD= (d-1>=0)? L[width*(influenceY+(d-1)*(length))+influenceX]: 8888;
					int nextD    = (d+1<maxDisparity)? L[width*(influenceY+(d+1)*(length))+influenceX] : 8888;

					//int minValue  = minArray(L,influenceX,influenceY,maxDisparity);
					int minValue =minimuns[influenceY*width+influenceX];
					int valueToAssign =  costPixel + minBetweenNumbersInt(currentD,nextD+p1,previousD+p1,minValue+p2) - minValue;

					L[width*(y+d*(length))+x] = valueToAssign;
					if(valueToAssign < minimunValue){
						minimunValue = valueToAssign;
					}

					/*if(valueToAssign <= 0){
						cout << x<<" "<<y<<"\t influence "<< influenceX << " " << influenceY << endl;
						cout << "Pixels: "<<x<<"-"<<y<<"\t"<<"Formula: " <<costPixel << "\t + min["<< currentD << "\t"<< previousD << "\t"
							<<nextD<<"\t"<<minValue<< "]" <<"\t Result Value: " <<valueToAssign<< "\t"<< d <<endl;
						waitKey(0);
					}*/
				}
				minimuns[y*width+x] = minimunValue;
			}
			x = x + increaseX;
		}
		y = y + increaseY;
	}


	cout << "done Aggregate Cost direction "<<directionx << " " << directiony << endl;
	free(minimuns);
}


__host__ int minBetweenNumbersInt(int a, int b, int c, int d){
	int min = a;

	if (min > b){
		min = b;
	}
	if (min > c){
		min = c;
	}
	if (min > d){
		min = d;
	}
	return min;
}


__host__ void DivideImagesCam(Mat completeImage, Mat* left, Mat* right){
	*left = completeImage(Rect(0,0,WIDTHIMAGE,LENGTHIMAGE));
	*right = completeImage(Rect(1280,0,WIDTHIMAGE,LENGTHIMAGE));
}

