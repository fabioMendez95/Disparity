#include <stdio.h>
#include <iostream>
#include <string.h>
#include <limits>

#include "opencv2/core/core.hpp"
#include <opencv2/imgproc/imgproc.hpp>
#include "opencv2/highgui/highgui.hpp"
#include <opencv2/contrib/contrib.hpp>
#include"string"

#include<sys/resource.h>
#include<time.h>
#include<sys/time.h>

#include<mutex>
#include<thread>

#include "Comparison.h"
#include "Kernels.cuh"
//#include "ZedCamera.h"
#include "Radar.h"
#include "Camera.h"

using namespace std;
using namespace cv;

#define threadx 16
#define thready 16

#define USERADAR true

#define SAVEIMAGE false
#define Profile true

#define DISPLAY true

#define USECAMARA true
#define Paths8 true
#define Camara 1
#define WIDTHIMAGE 1280
#define LENGTHIMAGE 720

__host__ void SGM();
__host__ Mat DisparityCreation(int* imageFromKernel, int width, int len);

__host__ String numberOfZeros(int number);
__host__ String getImageLocation(int frame, String side);

__host__ pointCoo getPoint(int ID, int pathX, int pathY, int width, int length);
__host__ void getKernelInitialInformation(int pathNumber, startInfo* pixelDi, int width, int length);
__host__ void radarThread();

bool readRadar = true;
mutex radarMtx;

int main (int argc, char** argv){
	//Comparison Calculations
	//Comparison comp;
	//comp.CompareDisparities();
	//------------------------

	SGM();
	return 0;
}

__host__ void radarThread(){
	/*bool correctlyRead = radar.readInfo();
	while (correctlyRead == 0) {
		correctlyRead = radar.readInfo();
	}*/
	Radar radar;
	radar.startRadar();
	bool copyReadRadar = true;
	while(copyReadRadar){
		radar.readInfo();
		radarMtx.lock();
		copyReadRadar = readRadar;
		radarMtx.unlock();
	}
	radar.closeRadar();
}

__host__ void SGM(){
	int frameNumber = 30;
	int frame = 0;
	Size size(720,400);
	cout << "Starting process \n";

	//Radar Info
#if USERADAR
	thread thrRadar(radarThread);
#endif
	struct timeval timstr;
	struct timeval timstrTotal;
	Mat left, right;
	Mat completeImage;
	//Texture Creation
	Mat leftD,rightD;
	uchar* imageLeftA,*imageRightA,*leftC,*rightC;


	//CensusDeclarations
	unsigned int* censusLa;
	unsigned int* censusRa;
	int* costK;


	//Maximum box value is depending on bytes used
	int BoxCostX = 9;
	int BoxCostY = 7;

	cout <<"Initialising camera\n";

#if USECAMARA
/*	ZedCamera zed;
	zed.initCamera();
	zed.grabImage();
	left = zed.getLeftImage();
	right = zed.getRightImage();*/
	Camera cam;
	cam.initCamera(1);
	cam.extractImage();
	right = cam.getRight();
	left = cam.getLeft();

#else
	//Reading Images
	left = imread("Images/KITTY/left/0000000000.png", CV_LOAD_IMAGE_COLOR);
	right= imread("Images/KITTY/right/0000000000.png", CV_LOAD_IMAGE_COLOR);
#endif

	cout << "Camera ready \n";
	//Initialisation Parameters---------------
	//First Kernel Params
	imageLeftA = (uchar*)malloc((sizeof(uchar))*(left.cols)*(left.rows));
	imageRightA= (uchar*)malloc((sizeof(uchar))*(right.cols)*(right.rows));

	int decreseX = BoxCostX/2 + BoxCostX/2;
	int decreseY = BoxCostY/2 + BoxCostY/2;

	int dimX = ((left.cols-decreseX) / threadx);
	int dimY = ((left.rows-decreseY) / thready);
	cout <<"Dimensions Grid: " << dimX << " " << dimY << endl;
	cout << "Dimensions Block: " << threadx << " " <<thready << endl;

	dim3 dimGrid(dimX,dimY);
	dim3 dimBlock(threadx,thready);
	//First Kernel, Census and cost Computation


	//Second Kernel Params
	int widthR = left.cols-decreseX;
	int lengthR = left.rows-decreseY;
	cout << "Disparity Size : " << widthR << " " << lengthR << endl;

#if USECAMARA && Paths8
	int threadNum = 382;
#elif USECAMARA
	int threadNum = 255;
#else
	int threadNum = 458; // pathNumber is divisible by this, 437 blocks
#endif

#if Paths8
	int pathNumber =(widthR+lengthR-1)*4 + widthR *2 + lengthR*2;
#else
	int pathNumber = widthR *2 + lengthR*2;
#endif
	dim3 dimGrid2(pathNumber/threadNum);
	dim3 dimBlock2(threadNum);
	//Assigning Paths

	//Setting Up initial Info, this is done just once in the algorithm.


	int* L1S = (int*) malloc((sizeof(int)) * (left.cols - decreseX) * (left.rows - decreseY)* (maxDisparity));
	//Done Initialisation Parameters----------


	startInfo* initialInfo, *initialInfoToKernel;
	initialInfo = (startInfo*) malloc((sizeof(startInfo)) * pathNumber);
	getKernelInitialInformation(pathNumber, initialInfo, widthR, lengthR);
	hipMalloc(&initialInfoToKernel, (sizeof(startInfo)) * pathNumber);
	hipMemcpy(initialInfoToKernel, initialInfo, (sizeof(startInfo)) * pathNumber, hipMemcpyHostToDevice);
	hipMalloc(&leftC, (sizeof(uchar)) * (left.cols) * (left.rows));
	hipMalloc(&rightC, (sizeof(uchar)) * (left.cols) * (left.rows));
	hipMalloc(&censusLa,(sizeof(unsigned int))*(left.cols-decreseX)*(left.rows-decreseY));
	hipMalloc(&censusRa,(sizeof(unsigned int))*(right.cols-decreseX)*(right.rows-decreseY));
	cout << "CUDA malloc 1: "<<hipMalloc(&costK,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity)) << endl;

	int* disKernel;
	int* disFromKernel = (int*) malloc((sizeof(int)) * (widthR) * (lengthR));
	int errorCUDAMALLOCdis = hipMalloc(&disKernel,(sizeof(int)*(widthR)*(lengthR)));
	cout << "Malloc dis: " << errorCUDAMALLOCdis << endl;
	//Timing Total
	gettimeofday(&timstrTotal, NULL);
	double beginTotal = timstrTotal.tv_sec + (timstrTotal.tv_usec / 1000000.0);


	int* L1;
	int errorCUDAMALLOC1 = hipMalloc(&L1,(sizeof(int)*(widthR)*(lengthR)*(maxDisparity)));
#if Profile
	cout <<"CUDA malloc " << errorCUDAMALLOC1 <<endl;
#endif

	cout << "\n\nStarting main loop \n";
	//--------------------------------------------------------------------------------------------------
	//----------------------------Main Loop-------------------------------------------------------------
	//--------------------------------------------------------------------------------------------------
	while (frame < frameNumber || USECAMARA) {
		//Stop Condition
		if (waitKey(1) >= 1) {
			cout << "Stoped at frame " << frame << endl;
			break;
		}
		//Read Image---------------------------------------
#if USECAMARA
/*		zed.grabImage();
		left = zed.getLeftImage();
		right = zed.getRightImage();*/
		cam.extractImage();
		right = cam.getRight();
		left = cam.getLeft();
#else
		left = imread(getImageLocation(frame,"left"), CV_LOAD_IMAGE_COLOR);
		right= imread(getImageLocation(frame,"right"), CV_LOAD_IMAGE_COLOR);
#endif
		//Done Read Image----------------------------------

#if Profile
		//Timing
		gettimeofday(&timstr, NULL);
		double begin = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
#endif
		//Converting Images--------------------------------
#if USECAMARA
/*		imag launching thread then read from iteLeftA = left.data;
		imageRightA = right.data;*/
		Mat leftBlack;
		cvtColor(left, leftBlack, CV_BGR2GRAY);
		Mat rightBlack;
		cvtColor(right, rightBlack, CV_BGR2GRAY);
		imageLeftA = leftBlack.data;
		imageRightA = rightBlack.data;
#if DISPLAY
		namedWindow("left");
		imshow("left",left);
#endif
#else
		Mat leftBlack;
		cvtColor(left, leftBlack, CV_BGR2GRAY);
		Mat rightBlack;
		cvtColor(right, rightBlack, CV_BGR2GRAY);
		imageLeftA = leftBlack.data;
		imageRightA = rightBlack.data;
#endif
		int errorMemCpy1 = hipMemcpy(leftC, imageLeftA, (sizeof(uchar)) * (left.cols) * (left.rows),hipMemcpyHostToDevice);
		int errorMemCpy2 = hipMemcpy(rightC, imageRightA,(sizeof(uchar)) * (left.cols) * (left.rows),hipMemcpyHostToDevice);
#if Profile
		cout<< "Cuda copy1: " << errorMemCpy1 << endl;
		cout<< "Cuda copy2: " << errorMemCpy2 << endl;
		//Done converting images---------------------------
#endif
		//------------First kernel cost Computation----------------------
		KernelDisparityCalculations<<<dimGrid,dimBlock>>>(BoxCostX,BoxCostY,censusLa,censusRa,widthR,lengthR,leftC,rightC,costK,L1,disKernel); //Old copy L1
		//------------Done-First-Kernel-----------------------------

		Mat disparity = DisparityCreation(disFromKernel,widthR,lengthR);
#if DISPLAY
		//Display Logic----------------------------------
		namedWindow("SMG");
		Mat display;
		disparity.convertTo(display, CV_8UC3, 255 / (maxDisparity), 0);
		applyColorMap(display,display,COLORMAP_HOT);
		resize(display,display,size);
		imshow("SMG", display);
		//Done Display Logic-----------------------------
#endif
		//----> Radar point fetch should be done here <---- TODO
#if USERADAR
		/*bool correctlyRead = radar.readInfo();
		while (correctlyRead == 0) {
			correctlyRead = radar.readInfo();
		}*/
#endif
		//-----------------------------------------------------



		//Second Kernel, Semi global matching and disparity Selection---------
		KernelSemiGlobal<<<dimGrid2,dimBlock2>>>(costK,widthR,lengthR,initialInfoToKernel,L1); //new Copy L1

		int syncStatus = hipDeviceSynchronize();
#if Profile
		if(syncStatus > 0){
			cout << "Error Kernel synchronise \t\tError type: " << syncStatus << "\n";
			cout << hipGetErrorString(hipGetLastError()) << endl;
		}
#endif
		//------------Done-Second-Kernel-----------------------------
		//----> Junction of information should be done here <---- TODO


		//int errorMemCpy3 = hipMemcpy(L1S, L1,(sizeof(int)) * (left.cols - decreseX) * (left.rows - decreseY)* (maxDisparity + 1), hipMemcpyDeviceToHost);
		int errorMemCpy3 = hipMemcpy(disFromKernel, disKernel,(sizeof(int)) * (widthR) * (lengthR), hipMemcpyDeviceToHost);
#if Profile
		cout<< "Cuda copy3: " << errorMemCpy3 << endl;
		cout << "Path Number " << pathNumber << endl;
#endif
		//Disparity Selection--------------------------------------------------------------------------------------------
		//Mat disparity = DisparitySelectionOneArray(L1S, widthR, lengthR);
		//Mat disparity(lengthR,widthR,CV_8U,disFromKernel);

		//---------------------------------------------------------------------------------------------------------------
#if Profile
		gettimeofday(&timstr, NULL);
		double end = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
		printf("Elapsed time Disparity:\t\t\t%.6lf (s)\n", end - begin);
#endif

#if SAVEIMAGE
		ostringstream imageSaveLocation;
		imageSaveLocation << "Results/"<<frame << ".png";
		imwrite(imageSaveLocation.str(),left);
#endif

		frame ++;
	}
	//--------------------------------------------------------------------------------------------------
	//----------------------------Main Loop-------------------------------------------------------------
	//--------------------------------------------------------------------------------------------------

	gettimeofday(&timstrTotal, NULL);
	double endTotal = timstrTotal.tv_sec + (timstrTotal.tv_usec / 1000000.0);
	printf("\n\nElapsed time:\t\t\t%.6lf (s)\n", endTotal - beginTotal);
	printf("Frames Analysed:\t\t%d frames\n",frame);
	printf("Frames per second:\t\t%f \n",frame/(endTotal-beginTotal));



#if USERADAR
	radarMtx.lock();
	readRadar = false;
	radarMtx.unlock();
	thrRadar.join();
#endif

#if USECAMARA
	//zed.closeCamera();
	cam.close();
#endif

	free(disFromKernel);
	free(initialInfo);
	free(L1S);
	hipDeviceReset();

}


__host__ pointCoo getPoint(int ID, int pathX, int pathY, int width, int length){
	pointCoo point;
	int startX;
	int startY;
	//Initial Point, this is the corner of the diagonals
	if(ID == 0){
		startX = 0;
		startY = 0;
		if(pathX == -1){
			startX = width - 1;
		}
		if(pathY == -1){
			startY = length - 1;
		}
	}
	//As diagonals go through the width and length, this needs to be divided into two segments
	//Segment 1 across the x axis of the image, fix y coordinate of the image
	else if (ID < width){
		startX = ID;
		startY = 0;
		if(pathX == -1){
			startX = startX - 1;
		}
		if(pathY == -1){
			startY = length-1;
		}
	}
	//Segment 2 across the y axis of the image, fix x coordinate
	else if (ID < width + length - 1){
		int newID = ID - width;
		startX = 0;
		startY = newID;
		if (pathX == -1){
			startX = width -1;
		}
		if (pathY == -1){
			startY = startY -1;
		}
	}

	point.x = startX;
	point.y = startY;
	return point;
}

__host__ void getKernelInitialInformation(int pathNumber, startInfo* pixelDi, int width, int length) {
	for (int blockID = 0; blockID < pathNumber; blockID++) {
		int ID = blockID;
		int LA = blockID; // Location in Array

		if (blockID < width) {
			pixelDi[LA].startX = ID;
			pixelDi[LA].startY = 0;
			pixelDi[LA].directionX = 0;
			pixelDi[LA].directionY = 1;
		} else if (blockID < 2 * width) {
			ID = blockID - width;
			pixelDi[LA].startX = ID;
			pixelDi[LA].startY = length - 1;
			pixelDi[LA].directionX = 0;
			pixelDi[LA].directionY = -1;
		} else if (blockID < 2 * width + length) {
			ID = blockID - 2 * width;
			pixelDi[LA].startX = 0;
			pixelDi[LA].startY = ID;
			pixelDi[LA].directionX = 1;
			pixelDi[LA].directionY = 0;
		} else if (blockID < 2 * width + 2 * length) {
			ID = blockID - 2 * width - length;
			pixelDi[LA].startX = width - 1;
			pixelDi[LA].startY = ID;
			pixelDi[LA].directionX = -1;
			pixelDi[LA].directionY = 0;
		}

		else if (blockID < 2 * width + 2 * length + (width + length - 1)) {
			//cout << "Should not happen \n";
			ID = blockID - 2 * width - 2 * length;
			pixelDi[LA].directionX = 1;
			pixelDi[LA].directionY = 1;
			pointCoo point = getPoint(ID, 1, 1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else if (blockID < 2 * width + 2 * length + 2 * (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length - (width + length - 1);
			pixelDi[LA].directionX = -1;
			pixelDi[LA].directionY = -1;
			pointCoo point = getPoint(ID, -1, -1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else if (blockID < 2 * width + 2 * length + 3 * (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length - 2 * (width + length - 1);
			pixelDi[LA].directionX = 1;
			pixelDi[LA].directionY = -1;
			pointCoo point = getPoint(ID, 1, -1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else if (blockID < 2 * width + 2 * length + 4 * (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length - 3 * (width + length - 1);
			pixelDi[LA].directionX = -1;
			pixelDi[LA].directionY = 1;
			pointCoo point = getPoint(ID, -1, 1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else {
			printf("ifs are wrong \n");
		}
	}
}

__host__ Mat DisparityCreation(int* imageFromKernel, int width, int len){
	Mat disparity(len,width,CV_8U);
	for(int y=0; y<len; y++){
		for(int x=0; x<width; x++){
			disparity.at<uchar>(y,x) = imageFromKernel[y*width+x];
		}
	}
	return disparity;
}

//side = left, right
__host__ String getImageLocation(int frame, String side){
	//"Images/KITTY/left/0000000000.png"
	String imageLocation = "Images/KITTY/"+side+"/"+numberOfZeros(frame);
	ostringstream convert;
	convert << frame;
	imageLocation = imageLocation + convert.str() +  ".png";
	return imageLocation;
}

__host__ String numberOfZeros(int number){
	if(number < 10){
		return "000000000";
	}
	else if (number < 100){
		return "00000000";
	}
	else if (number < 1000){
		return "0000000";
	}
	else{
		return "000000";
	}
}


