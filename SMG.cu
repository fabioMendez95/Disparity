#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <limits>

#include "opencv2/core/core.hpp"
#include <opencv2/imgproc/imgproc.hpp>
#include "opencv2/highgui/highgui.hpp"


#include<sys/resource.h>
#include<time.h>
#include<sys/time.h>


#include "Comparison.h"
#include "Kernels.cuh"
#include "ZedCamera.h"
#include "Radar.h"

using namespace std;
using namespace cv;

#define threadx 16
#define thready 16

#define USECAMARA false
#define Camara 1
#define WIDTHIMAGE 1280
#define LENGTHIMAGE 720

__host__ void SGM();
__host__ void AggregateCostCom(int* cost, int* L, int width, int length, int directionx, int directiony);
__host__ int minBetweenNumbersInt(int a, int b, int c, int d);
__host__ Mat DisparitySelectionP(int* L1, int* L2, int* L3, int* L4,/*int* L5, int* L6, int* L7, int* L8,*/ int width, int length);
__host__ void DivideImagesCam(Mat completeImage, Mat* left, Mat* right);
__host__ Mat DisparitySelectionOneArray(int*L, int width, int length);


__host__ pointCoo getPoint(int ID, int pathX, int pathY, int width, int length);
__host__ void getKernelInitialInformation(int pathNumber, startInfo* pixelDi, int width, int length);

//sl::Camera zed;

int main (int argc, char** argv){
	//Comparison Calculations
	//Comparison comp;
	//comp.CompareDisparities();
	//------------------------

	//Radar info
	Radar radar;
	radar.test();

	SGM();
	return 0;
}

__host__ void SGM(){
	struct timeval timstr;
	Mat left, right;
	Mat completeImage;
	//Texture Creation
	Mat leftD,rightD;
	uchar* imageLeftA,*imageRightA,*leftC,*rightC;
	startInfo* initialInfo, *initialInfoToKernel;

	//CensusDeclarations
	unsigned int* censusLa;
	unsigned int* censusRa;
	int* costK;
	int* L1;

	//Maximum box value is depending on bytes used
	int BoxCostX = 4;
	int BoxCostY = 2;
	ZedCamera zed;
#if USECAMARA
	zed.initCamera();

	char key = ' ';
	while(key != 'q'){
		zed.grabImage();
		left = zed.getLeftImage();
		right = zed.getRightImage();
		namedWindow("left");
		namedWindow("right");
		imshow("left", left);
		imshow("right", right);
		key = waitKey(10);
	}

	namedWindow("left");
	namedWindow("right");
	imshow("left", left);
	imshow("right", right);

#else
	//Reading Images
	left = imread("Images/KITTY/left/0000000000.png", CV_LOAD_IMAGE_COLOR);
	right= imread("Images/KITTY/right/0000000000.png", CV_LOAD_IMAGE_COLOR);

/*	left = imread("Images/Left.png", CV_LOAD_IMAGE_COLOR);
	right = imread("Images/Right.png", CV_LOAD_IMAGE_COLOR);*/

#endif

	//Initialisation Parameters---------------
	//First Kernel Params

	imageLeftA = (uchar*)malloc((sizeof(uchar))*(left.cols)*(left.rows));
	imageRightA= (uchar*)malloc((sizeof(uchar))*(right.cols)*(right.rows));

	int decreseX = BoxCostX/2 + BoxCostX/2;
	int decreseY = BoxCostY/2 + BoxCostY/2;

	int dimX = ((left.cols-decreseX) / threadx);
	int dimY = ((left.rows-decreseY) / thready);
	cout <<"Dimensions Grid: " << dimX << " " << dimY << endl;
	cout << "Dimensions Block: " << threadx << " " <<thready << endl;

	dim3 dimGrid(dimX,dimY);
	dim3 dimBlock(threadx,thready);
	//First Kernel, Census and cost Computation
	hipMalloc(&censusLa,(sizeof(unsigned int))*(left.cols-decreseX)*(left.rows-decreseY));
	hipMalloc(&censusRa,(sizeof(unsigned int))*(right.cols-decreseX)*(right.rows-decreseY));
	cout << "CUDA malloc 1: "<<hipMalloc(&costK,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1)) << endl;

	//Second Kernel Params
	int widthR = left.cols-decreseX;
	int lengthR = left.rows-decreseY;
	cout << "Disparity Size : " << widthR << " " << lengthR << endl;

	int threadNum = 258; // pathNumber is divisible by this, 437 blocks
	int pathNumber =/*(widthR+lengthR-1)*4 + */widthR *2 + lengthR*2;
	dim3 dimGrid2(pathNumber/threadNum);
	dim3 dimBlock2(threadNum);
	//Assigning Paths
	cout <<"CUDA malloc " <<hipMalloc(&L1,(sizeof(int)*(widthR)*(lengthR)*(maxDisparity+1)))<<endl;
	//Setting Up initial Info, this is done just once in the algorithm.
	initialInfo = (startInfo*)malloc((sizeof(startInfo))*pathNumber);
	getKernelInitialInformation(pathNumber,initialInfo,widthR,lengthR);
	hipMalloc(&initialInfoToKernel,(sizeof(startInfo))*pathNumber);
	hipMemcpy(initialInfoToKernel,initialInfo,(sizeof(startInfo))*pathNumber,hipMemcpyHostToDevice);
	//Done Initialisation Parameters----------

	//Loop should start here---------------------------------------------------------------------------------
	//Timing
	gettimeofday(&timstr, NULL);
	double begin = timstr.tv_sec + (timstr.tv_usec / 1000000.0);

	//Converting Images
#if USECAMARA
//	cout << zed.type2str(left.type()) << endl;
/*	Mat leftBlack;
	left.copyTo(leftBlack);
	Mat rightBlack;
	right.copyTo(rightBlack);*/
	namedWindow("leftB");
	namedWindow("rightB");
	//This line is wrong, not showing the correct image
	imshow("leftB", left);
	imshow("rightB", right);

	imageLeftA = left.data;
	imageRightA = right.data;
#else
	Mat leftBlack;
	cvtColor(left, leftBlack, CV_BGR2GRAY);
	Mat rightBlack;
	cvtColor(right, rightBlack, CV_BGR2GRAY);
	cout << zed.type2str(leftBlack.type()) << endl;
	imageLeftA = leftBlack.data;
	imageRightA= rightBlack.data;
#endif
	hipMalloc(&leftC,(sizeof(uchar))*(left.cols)*(left.rows));
	hipMalloc(&rightC,(sizeof(uchar))*(left.cols)*(left.rows));
	hipMemcpy(leftC,imageLeftA,(sizeof(uchar))*(left.cols)*(left.rows),hipMemcpyHostToDevice);
	hipMemcpy(rightC,imageRightA,(sizeof(uchar))*(left.cols)*(left.rows),hipMemcpyHostToDevice);
	//Done COnverting Images


	gettimeofday(&timstr, NULL);
	double begin1 = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
	//CensusAndCostKernel<<<dimGrid,dimBlock>>>(BoxCostX,BoxCostY,widthR,lengthR,leftC,rightC,costK); //Not running
	KernelDisparityCalculations<<<dimGrid,dimBlock>>>(BoxCostX,BoxCostY,censusLa,censusRa,left.cols-decreseX,left.rows-decreseY,leftC,rightC,costK);
	cout << "Synchronise status kernel 1: "<<hipDeviceSynchronize() << "\n";
	//------------Done-First-Kernel-----------------------------
	gettimeofday(&timstr, NULL);
	double end1 = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
	printf("Elapsed time Census and Cost:\t\t\t%.6lf (s)\n", end1 - begin1);


	gettimeofday(&timstr, NULL);
	double begin2 = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
	//Second Kernel, Semi global matching and disparity Selection

	KernelSemiGlobal<<<dimGrid2,dimBlock2>>>(costK,widthR,lengthR,initialInfoToKernel,L1);
	cout << "Synchronise status kernel 2: "<<hipDeviceSynchronize() << "\n";
	//------------Done-Second-Kernel-----------------------------

	gettimeofday(&timstr, NULL);
	double end2 = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
	printf("Elapsed time Aggregate Cost:\t\t\t%.6lf (s)\n", end2 - begin2);

	cout << "Done CUDA " << endl;
	int* L1S = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	hipMemcpy(L1S,L1,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1),hipMemcpyDeviceToHost);

	cout << "Path Number " << pathNumber << endl;


	//Disparity Selection--------------------------------------------------------------------------------------------
	Mat disparity = DisparitySelectionOneArray(L1S,widthR,lengthR);

	//---------------------------------------------------------------------------------------------------------------

	gettimeofday(&timstr, NULL);
	double end = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
	printf("Elapsed time Disparity:\t\t\t%.6lf (s)\n", end - begin);

	namedWindow("SMG");
	imshow("SMG",disparity);
	//imwrite("disparity.png",disparity);
	waitKey(0);
	//Loop should end here-------------------------------------------------------------------------------------------

	//free Memory
	hipFree(leftC);
	hipFree(rightC);
	hipFree(censusLa);
	hipFree(censusRa);
	hipFree(costK);
	hipFree(L1);
	hipFree(initialInfoToKernel);
	free(L1S);
	free(initialInfo);

#if USECAMARA
	zed.closeCamera();
#endif
}


__host__ pointCoo getPoint(int ID, int pathX, int pathY, int width, int length){
	pointCoo point;
	int startX;
	int startY;
	//Initial Point, this is the corner of the diagonals
	if(ID == 0){
		startX = 0;
		startY = 0;
		if(pathX == -1){
			startX = width - 1;
		}
		if(pathY == -1){
			startY = length - 1;
		}
	}
	//As diagonals go through the width and length, this needs to be divided into two segments
	//Segment 1 across the x axis of the image, fix y coordinate of the image
	else if (ID < width){
		startX = ID;
		startY = 0;
		if(pathX == -1){
			startX = startX - 1;
		}
		if(pathY == -1){
			startY = length-1;
		}
	}
	//Segment 2 across the y axis of the image, fix x coordinate
	else if (ID < width + length - 1){
		int newID = ID - width;
		startX = 0;
		startY = newID;
		if (pathX == -1){
			startX = width -1;
		}
		if (pathY == -1){
			startY = startY -1;
		}
	}

	point.x = startX;
	point.y = startY;
	return point;
}

__host__ void getKernelInitialInformation(int pathNumber, startInfo* pixelDi, int width, int length) {
	for (int blockID = 0; blockID < pathNumber; blockID++) {
		int ID = blockID;
		int LA = blockID; // Location in Array

		if (blockID < width) {
			pixelDi[LA].startX = ID;
			pixelDi[LA].startY = 0;
			pixelDi[LA].directionX = 0;
			pixelDi[LA].directionY = 1;
		} else if (blockID < 2 * width) {
			ID = blockID - width;
			pixelDi[LA].startX = ID;
			pixelDi[LA].startY = length - 1;
			pixelDi[LA].directionX = 0;
			pixelDi[LA].directionY = -1;
		} else if (blockID < 2 * width + length) {
			ID = blockID - 2 * width;
			pixelDi[LA].startX = 0;
			pixelDi[LA].startY = ID;
			pixelDi[LA].directionX = 1;
			pixelDi[LA].directionY = 0;
		} else if (blockID < 2 * width + 2 * length) {
			ID = blockID - 2 * width - length;
			pixelDi[LA].startX = width - 1;
			pixelDi[LA].startY = ID;
			pixelDi[LA].directionX = -1;
			pixelDi[LA].directionY = 0;
		} else if (blockID < 2 * width + 2 * length + (width + length - 1)) {
			cout << "Should not happen \n";
			ID = blockID - 2 * width - 2 * length;
			pixelDi[LA].directionX = 1;
			pixelDi[LA].directionY = 1;
			pointCoo point = getPoint(ID, 1, 1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else if (blockID < 2 * width + 2 * length + 2 * (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length - (width + length - 1);
			pixelDi[LA].directionX = -1;
			pixelDi[LA].directionY = -1;
			pointCoo point = getPoint(ID, -1, -1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else if (blockID < 2 * width + 2 * length + 3 * (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length - 2 * (width + length - 1);
			pixelDi[LA].directionX = 1;
			pixelDi[LA].directionY = -1;
			pointCoo point = getPoint(ID, 1, -1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else if (blockID < 2 * width + 2 * length + 4 * (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length - 3 * (width + length - 1);
			pixelDi[LA].directionX = -1;
			pixelDi[LA].directionY = 1;
			pointCoo point = getPoint(ID, -1, 1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else {
			printf("ifs are wrong \n");
		}
	}
}


//Disparity Selection Process
__host__ Mat DisparitySelectionP(int* L1, int* L2, int* L3, int* L4/*,int* L5, int* L6, int* L7, int* L8*/, int width, int length){
	Mat disparity(length,width,CV_8U);

	for(int y=0; y<length; y++){
		for(int x=0; x<width; x++){
			int costA = 99999;
			int disPix = 0;
			for (int d = 0; d<maxDisparity; d++){
				int sumAgg = L1[width*(y+d*length)+x] + L2[width*(y+d*(length))+x]+
						L3[width*(y+d*length)+x] + L4[width*(y+d*(length))+x] /*+ L5[width*(y+d*(length))+x]+
						L6[width*(y+d*length)+x] + L7[width*(y+d*(length))+x] + L8[width*(y+d*length)+x]*/;
				if(x == 100 && y == 369 && d == 99){
					cout << x << " " << y<< " : "<<L1[width*(y+d*length)+x] << endl;
					cout << width << " " << length << endl;
				}
				if (sumAgg < costA){
					costA = sumAgg;
					disPix = d;
				}
 			}
			//cout << "disparity pixel " <<x << " " << y << " is " << disPix << " with value "<< costA<< endl;
			disparity.at<uchar>(y,x) = disPix;
		}
	}
	return disparity;
}

__host__ Mat DisparitySelectionOneArray(int*L, int width, int length){
	Mat disparity(length,width,CV_8U);
	for(int y=0;y<length;y++){
		for(int x=0;x<width;x++){
			int costA = 99999;
			int disPix = 0;
			for (int d=0;d<maxDisparity;d++){
				int value = L[width*(y+d*length)+x];
				if(value < costA){
					costA = value;
					disPix = d;
				}
			}
			disparity.at<uchar>(y,x) = disPix;
		}
	}
	return disparity;
}

//Gets the cost Computation Across all Paths
__host__ void AggregateCostCom(int* cost, int* L, int width, int length, int directionx, int directiony){
	//penalties
/*	int p1 = 10;
	int p2 = 100;*/

	int startX,startY, increaseX, increaseY;

	if(directionx <= 0){
		startX = 0;
		increaseX = 1;
	}
	else{
		startX = width-1;
		increaseX = -1;
	}
	if(directiony <= 0){
		startY = 0;
		increaseY = 1;
	}
	else{
		startY = length-1;
		increaseY = -1;
	}

	int* minimuns = (int*)malloc((sizeof(int))*(width)*(length));
	int x;
	int y = startY;

	for(int yC=0; yC<length; yC++){
		int influenceY = y + directiony;
		x = startX;
		for(int xC=0; xC<width; xC++){
			int influenceX = x + directionx;

			int minimunValue = 999999;

			if(influenceX > width || influenceX < 0 || influenceY > length || influenceY < 0){
				for (int d = 0; d<maxDisparity; d++){
					int costPixel = cost[width*(y+d*(length))+x];
					L[width*(y+d*(length))+x] = costPixel;
					if(costPixel < minimunValue){
						minimunValue = costPixel;
						//cout << "minimum ->  " << costPixel;
					}
				}
			}
			else{
				/*cout << x << " " << y <<endl;
				waitKey(0);*/
				for (int d = 0;  d<maxDisparity; d++){
					int costPixel = cost[width*(y+d*(length))+x];
					int currentD = L[width*(influenceY+d*(length))+influenceX];
					int previousD= (d-1>=0)? L[width*(influenceY+(d-1)*(length))+influenceX]: 8888;
					int nextD    = (d+1<maxDisparity)? L[width*(influenceY+(d+1)*(length))+influenceX] : 8888;

					//int minValue  = minArray(L,influenceX,influenceY,maxDisparity);
					int minValue =minimuns[influenceY*width+influenceX];
					int valueToAssign =  costPixel + minBetweenNumbersInt(currentD,nextD+p1,previousD+p1,minValue+p2) - minValue;

					L[width*(y+d*(length))+x] = valueToAssign;
					if(valueToAssign < minimunValue){
						minimunValue = valueToAssign;
					}

					/*if(valueToAssign <= 0){
						cout << x<<" "<<y<<"\t influence "<< influenceX << " " << influenceY << endl;
						cout << "Pixels: "<<x<<"-"<<y<<"\t"<<"Formula: " <<costPixel << "\t + min["<< currentD << "\t"<< previousD << "\t"
							<<nextD<<"\t"<<minValue<< "]" <<"\t Result Value: " <<valueToAssign<< "\t"<< d <<endl;
						waitKey(0);
					}*/
				}
				minimuns[y*width+x] = minimunValue;
			}
			x = x + increaseX;
		}
		y = y + increaseY;
	}


	cout << "done Aggregate Cost direction "<<directionx << " " << directiony << endl;
	free(minimuns);
}


__host__ int minBetweenNumbersInt(int a, int b, int c, int d){
	int min = a;

	if (min > b){
		min = b;
	}
	if (min > c){
		min = c;
	}
	if (min > d){
		min = d;
	}
	return min;
}


__host__ void DivideImagesCam(Mat completeImage, Mat* left, Mat* right){
	*left = completeImage(Rect(0,0,WIDTHIMAGE,LENGTHIMAGE));
	*right = completeImage(Rect(1280,0,WIDTHIMAGE,LENGTHIMAGE));
}

