#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <limits>
//#include "ReadRadar.h"

#include "opencv2/core/core.hpp"
#include <opencv2/imgproc/imgproc.hpp>
#include "opencv2/highgui/highgui.hpp"
#include <sl/Camera.hpp>

#include<sys/resource.h>
#include <time.h>
#include<sys/time.h>

#include "Comparison.h"
#include "Kernels.cuh"

using namespace std;
using namespace cv;

#define threadx 16
#define thready 16

#define USECAMARA false
#define Camara 1
#define WIDTHIMAGE 1280
#define LENGTHIMAGE 720

__host__ void SMG();
__host__ void AggregateCostCom(int* cost, int* L, int width, int length,int maxDisparity, int directionx, int directiony);
__host__ int minBetweenNumbersInt(int a, int b, int c, int d);
__host__ Mat DisparitySelectionP(int* L2, int* L4, int* L5, int* L7,int* L1, int* L3, int* L6, int* L8, int maxDisparity, int width, int length);
__host__ void DivideImagesCam(Mat completeImage, Mat* left, Mat* right);
__host__ void CensusTransformation (Mat image, int widthW, int lengthW, unsigned int* censusArray);
__host__ void CostComputationCensus (unsigned int* censusL, unsigned int* censusR, int* cost, int maxDisparity, int width, int length);
__host__ int HammingDistanceNumbers (unsigned int a, unsigned int b);


//sl::Camera zed;

int main (int argc, char** argv){
	//Initialising Radar
	//ReadRadar radar;
	//radar.Connect();
	//radar.Read();
	//Initialising Radar


	//Comparison Calculations
	//Comparison comp;
	//comp.CompareDisparities();
	//------------------------

	SMG();
	return 0;
}

__host__ void SMG(){
	struct timeval timstr;
	Mat left, right;
	Mat completeImage;
	//Texture Creation
	Mat leftD,rightD;
	hipArray* leftImageCUDA;
	hipArray* rightImageCUDA;
	hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	//Maximum box value is depending on bytes used
	int BoxCostX = 9;
	int BoxCostY = 7;
	int maxDisparity = 100;


#if USECAMARA
	VideoCapture stream(Camara);
	for (int i =0; i<100;i++){
		stream.read(completeImage);
	}

	DivideImagesCam(completeImage,&right,&left);
#else
	//Reading Images
	left = imread("Images/KITTY/left/0000000000.png", CV_LOAD_IMAGE_COLOR);
	right= imread("Images/KITTY/right/0000000000.png", CV_LOAD_IMAGE_COLOR);

#endif

	//Converting Images
	Mat leftBlack;
	cvtColor( left, leftBlack, CV_BGR2GRAY );
	Mat rightBlack;
	cvtColor( right, rightBlack, CV_BGR2GRAY );

	//Done Converting Images

	int decreseX = BoxCostX/2 + BoxCostX/2;
	int decreseY = BoxCostY/2 + BoxCostY/2;

	int dimX = (leftBlack.cols-decreseX) / threadx;
	int dimY = (leftBlack.rows-decreseY) / thready;
	dim3 dimGrid(dimX,dimY);
	dim3 dimBlock(threadx,thready);


	//Timing
	gettimeofday(&timstr, NULL);
	double begin = timstr.tv_sec + (timstr.tv_usec / 1000000.0);

	//Creating Textures CUDA
	hipMallocArray(&leftImageCUDA, &channelDesc1,leftBlack.cols,leftBlack.rows);
	hipMallocArray(&rightImageCUDA, &channelDesc2, rightBlack.cols,rightBlack.rows);

	leftBlack.convertTo(leftD,CV_32F);
	rightBlack.convertTo(rightD,CV_32F);

	//cout << (int)leftD.at<uchar>(10,10) << " " << sizeof(float)<<endl;

	hipMemcpyToArray(leftImageCUDA, 0, 0, leftD.data, ((sizeof(float))*leftD.cols*leftD.rows),hipMemcpyHostToDevice);
	hipMemcpyToArray(rightImageCUDA,0, 0,rightD.data,((sizeof(float))*rightD.cols*rightD.rows),hipMemcpyHostToDevice);
	ImageLeftTex.addressMode[0] = hipAddressModeWrap;
	ImageLeftTex.addressMode[1] = hipAddressModeWrap;
	ImageLeftTex.filterMode = hipFilterModeLinear;
	ImageLeftTex.normalized = false;
	ImageRightTex.addressMode[0] = hipAddressModeWrap;
	ImageRightTex.addressMode[1] = hipAddressModeWrap;
	ImageRightTex.filterMode = hipFilterModeLinear;
	ImageRightTex.normalized = false;
	hipBindTextureToArray(ImageLeftTex, leftImageCUDA,channelDesc1);
	hipBindTextureToArray(ImageRightTex, rightImageCUDA, channelDesc2);
	//DONE creating textures



	//Census Transform
	unsigned int* censusLa = (unsigned int*)malloc((sizeof(unsigned int))*(left.cols-decreseX)*(left.rows-decreseY));
	unsigned int* censusRa = (unsigned int*)malloc((sizeof(unsigned int))*(right.cols-decreseX)*(right.rows-decreseY));
	CensusTransformation(leftBlack,BoxCostX,BoxCostY,censusLa);
	CensusTransformation(rightBlack,BoxCostX,BoxCostY,censusRa);

	hipMalloc(&censusLa,(sizeof(unsigned int))*(left.cols-decreseX)*(left.rows-decreseY));
	hipMalloc(&censusRa,(sizeof(unsigned int))*(right.cols-decreseX)*(right.rows-decreseY));

	KernelDisparityCalculations<<<dimGrid,dimBlock>>>(BoxCostX,BoxCostY,censusLa,censusRa);
	hipDeviceSynchronize();



	cout << "census ok" << endl;
	cout << "Image size: " << leftBlack.cols << " " << leftBlack.rows << endl;

	//Cost Computation
	int* cost = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	//CostComputation(leftBlack,rightBlack,cost,maxDisparity,left.cols,left.rows,BoxCostX,BoxCostY);
	CostComputationCensus(censusLa,censusRa,cost,maxDisparity,left.cols-decreseX,left.rows-decreseY);
	cout << "cost ok" << endl;


	//Aggregate Cost
	int* L1 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L1,left.cols-decreseX,left.rows-decreseY,maxDisparity,-1,-1);
	cout << "Done" << endl;

	int* L2 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L2,left.cols-decreseX,left.rows-decreseY,maxDisparity,0,-1);
	cout << "Done" << endl;

	int* L3 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L3,left.cols-decreseX,left.rows-decreseY,maxDisparity,1,-1);
	cout << "Done" << endl;

	int* L4 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L4,left.cols-decreseX,left.rows-decreseY,maxDisparity,-1,0);
	cout << "Done" << endl;

	int* L5 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L5,left.cols-decreseX,left.rows-decreseY,maxDisparity,1,0);
	cout << "Done" << endl;

	int* L6 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L6,left.cols-decreseX,left.rows-decreseY,maxDisparity,-1,1);
	cout << "Done" << endl;

	int* L7 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L7,left.cols-decreseX,left.rows-decreseY,maxDisparity,0,1);
	cout << "Done" << endl;

	int* L8 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L8,left.cols-decreseX,left.rows-decreseY,maxDisparity,1,1);
	cout << "Done" << endl;




	//Disparity Selection
	Mat disparity = DisparitySelectionP(L1,L2,L3,L4,L5,L6,L7,L8,maxDisparity,left.cols-decreseX,left.rows-decreseY);
	gettimeofday(&timstr, NULL);
	double end = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
	printf("Elapsed time Local Matching:\t\t\t%.6lf (s)\n", end - begin);


	namedWindow("SMG");
	imshow("SMG",disparity);
	imwrite("disparity.png",disparity);
	waitKey(0);


	//free Memory
	hipUnbindTexture(ImageLeftTex);
	hipUnbindTexture(ImageRightTex);
	hipFree(censusLa);
	hipFree(censusRa);
	free(censusLa);
	free(censusRa);
	free(cost);
	free(L1);
	free(L2);
	free(L3);
	free(L4);
	free(L5);
	free(L6);
	free(L7);
	free(L8);
}
//Disparity Selection Process
__host__ Mat DisparitySelectionP(int* L2, int* L4, int* L5, int* L7,int* L1, int* L3, int* L6, int* L8, int maxDisparity, int width, int length){
	Mat disparity(length,width,CV_8U);

	for(int y=0; y<length; y++){
		for(int x=0; x<width; x++){
			int costA = 99999;
			int disPix = 0;
			for (int d = 0; d<maxDisparity; d++){
				int sumAgg = L1[width*(y+d*length)+x] + L2[width*(y+d*(length))+x]+
						L3[width*(y+d*length)+x] + L4[width*(y+d*(length))+x] + L5[width*(y+d*(length))+x]+
						L6[width*(y+d*length)+x] + L7[width*(y+d*(length))+x] + L8[width*(y+d*length)+x];
				if (sumAgg < costA){
					costA = sumAgg;
					disPix = d;
				}
 			}
			//cout << "disparity pixel " <<x << " " << y << " is " << disPix << " with value "<< costA<< endl;
			disparity.at<uchar>(y,x) = disPix;
		}
	}
	return disparity;
}


//Gets the cost Computation Across all Paths
__host__ void AggregateCostCom(int* cost, int* L, int width, int length, int maxDisparity, int directionx, int directiony){
	//penalties
	int p1 = 5;
	int p2 = 100;

	int startX,startY, increaseX, increaseY;

	if(directionx <= 0){
		startX = 0;
		increaseX = 1;
	}
	else{
		startX = width-1;
		increaseX = -1;
	}
	if(directiony <= 0){
		startY = 0;
		increaseY = 1;
	}
	else{
		startY = length-1;
		increaseY = -1;
	}

	int* minimuns = (int*)malloc((sizeof(int))*(width)*(length));
	int x;
	int y = startY;

	for(int yC=0; yC<length; yC++){
		int influenceY = y + directiony;
		x = startX;
		for(int xC=0; xC<width; xC++){
			int influenceX = x + directionx;

			int minimunValue = 999999;

			if(influenceX > width || influenceX < 0 || influenceY > length || influenceY < 0){
				for (int d = 0; d<maxDisparity; d++){
					int costPixel = cost[width*(y+d*(length))+x];
					L[width*(y+d*(length))+x] = costPixel;
					if(costPixel < minimunValue){
						minimunValue = costPixel;
						//cout << "minimum ->  " << costPixel;
					}
				}
			}
			else{
				/*cout << x << " " << y <<endl;
				waitKey(0);*/
				for (int d = 0;  d<maxDisparity; d++){
					int costPixel = cost[width*(y+d*(length))+x];
					int currentD = L[width*(influenceY+d*(length))+influenceX];
					int previousD= (d-1>=0)? L[width*(influenceY+(d-1)*(length))+influenceX]: 8888;
					int nextD    = (d+1<maxDisparity)? L[width*(influenceY+(d+1)*(length))+influenceX] : 8888;

					//int minValue  = minArray(L,influenceX,influenceY,maxDisparity);
					int minValue =minimuns[influenceY*width+influenceX];
					int valueToAssign =  costPixel + minBetweenNumbersInt(currentD,nextD+p1,previousD+p1,minValue+p2) - minValue;

					L[width*(y+d*(length))+x] = valueToAssign;
					if(valueToAssign < minimunValue){
						minimunValue = valueToAssign;
					}

					/*if(valueToAssign <= 0){
						cout << x<<" "<<y<<"\t influence "<< influenceX << " " << influenceY << endl;
						cout << "Pixels: "<<x<<"-"<<y<<"\t"<<"Formula: " <<costPixel << "\t + min["<< currentD << "\t"<< previousD << "\t"
							<<nextD<<"\t"<<minValue<< "]" <<"\t Result Value: " <<valueToAssign<< "\t"<< d <<endl;
						waitKey(0);
					}*/
				}
				minimuns[y*width+x] = minimunValue;
			}
			x = x + increaseX;
		}
		y = y + increaseY;
	}


	cout << "done Aggregate Cost direction "<<directionx << " " << directiony << endl;
	free(minimuns);
}


__host__ int minBetweenNumbersInt(int a, int b, int c, int d){
	int min = a;

	if (min > b){
		min = b;
	}
	if (min > c){
		min = c;
	}
	if (min > d){
		min = d;
	}
	return min;
}


//Testing Census---------------
__host__ void CensusTransformation (Mat image, int widthW, int lengthW, unsigned int* censusArray){
	unsigned int census = 0;
	int shiftCount = 0;

	int width = image.cols - widthW/2 - widthW/2;

	for (int y = lengthW / 2; y < image.rows - lengthW / 2; y++) {
		for (int x = widthW / 2; x < image.cols - widthW / 2; x++) {
			census = 0;
			shiftCount = 0;
			int xA = x - widthW / 2;
			int yA = y - lengthW / 2;

			for (int j = y - lengthW / 2; j <= y + lengthW / 2; j++) {
				for (int i = x - widthW / 2; i <= x + widthW / 2; i++) {
					if ((int) image.at<uchar>(y, x) < (int) image.at<uchar>(j, i) && shiftCount != widthW * lengthW / 2) {
						census <<= 1;
						census = census + 1;
					} else if (shiftCount != widthW * lengthW / 2) {
						census <<= 1;
					}
					shiftCount++;
				}
			}
			censusArray[yA * width + xA] = census;
		}
	}
}

__host__ void CostComputationCensus (unsigned int* censusL, unsigned int* censusR, int* cost, int maxDisparity, int width, int length){
	for(int y=0;y<length;y++){
		for(int xl=0;xl<width;xl++){
			int start = xl-(maxDisparity);

			unsigned int valueLeft = censusL[y*width+xl];
			//cout << "Value Left " << valueLeft <<endl;
			for(int xr = start; xr<=xl; xr++){
				int valueToAssigned;
				int dis = xl-xr;
				//cout << dis<<" "<<xr << endl;
				if(xr>=0){
					unsigned int valueRight = censusR[y*width+xr];
					//cout << "Value Right " << valueRight << endl;
					valueToAssigned = HammingDistanceNumbers(valueLeft,valueRight);
				}
				else{
					valueToAssigned = 99999;
				}
				cost[width*(y+dis*(length))+xl] =valueToAssigned;

			}
		}
	}
}

__host__ int HammingDistanceNumbers (unsigned int a, unsigned int b){
	unsigned int val = a ^ b;
	int dist = 0;
	while(val != 0){
		val = val & (val-1);
		dist++;
	}
	return dist;
}


__host__ void DivideImagesCam(Mat completeImage, Mat* left, Mat* right){
	*left = completeImage(Rect(0,0,WIDTHIMAGE,LENGTHIMAGE));
	*right = completeImage(Rect(1280,0,WIDTHIMAGE,LENGTHIMAGE));
}

