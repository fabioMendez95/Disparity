#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <limits>
//#include "ReadRadar.h"

#include "opencv2/core/core.hpp"
#include <opencv2/imgproc/imgproc.hpp>
#include "opencv2/highgui/highgui.hpp"
#include <sl/Camera.hpp>

#include<sys/resource.h>
#include <time.h>
#include<sys/time.h>

#include "Comparison.h"
#include "Kernels.cuh"

using namespace std;
using namespace cv;

#define threadx 16
#define thready 16

#define USECAMARA false
#define Camara 1
#define WIDTHIMAGE 1280
#define LENGTHIMAGE 720

__host__ void SMG();
__host__ void AggregateCostCom(int* cost, int* L, int width, int length,int maxDisparity, int directionx, int directiony);
__host__ int minBetweenNumbersInt(int a, int b, int c, int d);
__host__ Mat DisparitySelectionP(int* L1, int* L2, int* L3, int* L4,/*int* L5, int* L6, int* L7, int* L8,*/ int maxDisparity, int width, int length);
__host__ void DivideImagesCam(Mat completeImage, Mat* left, Mat* right);

__host__ pointCoo getPoint(int ID, int pathX, int pathY, int width, int length);
__host__ void getKernelInitialInformation(int pathNumber, startInfo* pixelDi, int width, int length);

//sl::Camera zed;

int main (int argc, char** argv){
	//Initialising Radar
	//ReadRadar radar;
	//radar.Connect();
	//radar.Read();
	//Initialising Radar


	//Comparison Calculations
	//Comparison comp;
	//comp.CompareDisparities();
	//------------------------

	SMG();
	return 0;
}

__host__ void SMG(){
	struct timeval timstr;
	Mat left, right;
	Mat completeImage;
	//Texture Creation
	Mat leftD,rightD;
	uchar* imageLeftA,*imageRightA,*leftC,*rightC;
	startInfo* initialInfo, *initialInfoToKernel;

	//CensusDeclarations
	unsigned int* censusLa;
	unsigned int* censusRa;
	int* costK;
	int* L1;
	int* L2;
	int* L3;
	int* L4;
	/*int* L5;
	int* L6;
	int* L7;
	int* L8;*/

	//Maximum box value is depending on bytes used
	int BoxCostX = 9;
	int BoxCostY = 7;
	int maxDisparity = 100;


#if USECAMARA
	VideoCapture stream(Camara);
	for (int i =0; i<100;i++){
		stream.read(completeImage);
	}

	DivideImagesCam(completeImage,&right,&left);
#else
	//Reading Images
	left = imread("Images/KITTY/left/0000000000.png", CV_LOAD_IMAGE_COLOR);
	right= imread("Images/KITTY/right/0000000000.png", CV_LOAD_IMAGE_COLOR);

#endif

	//Converting Images
	Mat leftBlack;
	cvtColor( left, leftBlack, CV_BGR2GRAY );
	Mat rightBlack;
	cvtColor( right, rightBlack, CV_BGR2GRAY );

	//Passing images to kernels----------------------------
	imageLeftA = (uchar*)malloc((sizeof(uchar))*(left.cols)*(left.rows));
	imageRightA= (uchar*)malloc((sizeof(uchar))*(right.cols)*(right.rows));
	imageLeftA = leftBlack.data;
	imageRightA= rightBlack.data;
	hipMalloc(&leftC,(sizeof(uchar))*(left.cols)*(left.rows));
	hipMalloc(&rightC,(sizeof(uchar))*(left.cols)*(left.rows));
	hipMemcpy(leftC,imageLeftA,(sizeof(uchar))*(left.cols)*(left.rows),hipMemcpyHostToDevice);
	hipMemcpy(rightC,imageRightA,(sizeof(uchar))*(left.cols)*(left.rows),hipMemcpyHostToDevice);
	//Done passing images to kernels------------------------

	int decreseX = BoxCostX/2 + BoxCostX/2;
	int decreseY = BoxCostY/2 + BoxCostY/2;

	int dimX = ((leftBlack.cols-decreseX) / threadx);
	int dimY = ((leftBlack.rows-decreseY) / thready);
	dim3 dimGrid(dimX,dimY);
	dim3 dimBlock(threadx,thready);

	//Timing
	gettimeofday(&timstr, NULL);
	double begin = timstr.tv_sec + (timstr.tv_usec / 1000000.0);

	//First Kernel, Census and cost Computation
	hipMalloc(&censusLa,(sizeof(unsigned int))*(left.cols-decreseX)*(left.rows-decreseY));
	hipMalloc(&censusRa,(sizeof(unsigned int))*(right.cols-decreseX)*(right.rows-decreseY));
	hipMalloc(&costK,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));

	KernelDisparityCalculations<<<dimGrid,dimBlock>>>(BoxCostX,BoxCostY,censusLa,censusRa,left.cols-decreseX,left.rows-decreseY,leftC,rightC,costK);
	cout << "Synchronise status: "<<hipDeviceSynchronize() << endl;

	//------------Done-First-Kernel-----------------------------


	//Second Kernel, Semi global matching and disparity Selection
	int widthR = left.cols-decreseX;
	int lengthR = left.rows-decreseY;
	int threadNum = 229; // pathNumber is divisible by this, 437 blocks
	int pathNumber =/*(widthR+lengthR-1)*4 + */widthR *2 + lengthR*2;
	dim3 dimGrid2(pathNumber/threadNum);
	dim3 dimBlock2(threadNum);
	//Assigning Paths
	cout <<"CUDA malloc " <<hipMalloc(&L1,(sizeof(int)*(widthR)*(lengthR)*(maxDisparity+1)))<<endl;
	cout <<"CUDA malloc " <<hipMalloc(&L2,(sizeof(int)*(widthR)*(lengthR)*(maxDisparity+1)))<<endl;
	cout <<"CUDA malloc " <<hipMalloc(&L3,(sizeof(int)*(widthR)*(lengthR)*(maxDisparity+1)))<<endl;
	cout <<"CUDA malloc " <<hipMalloc(&L4,(sizeof(int)*(widthR)*(lengthR)*(maxDisparity+1)))<<endl;
	//cout <<"this new " <<hipMalloc(&L5,(sizeof(int)*(widthR)*(lengthR)*(maxDisparity+1)))<<endl;
	//cout <<"this new " <<hipMalloc(&L6,(sizeof(int)*(widthR)*(lengthR)*(maxDisparity+1)))<<endl;
	//cout <<"this new " <<hipMalloc(&L7,(sizeof(int)*(widthR)*(lengthR)*(maxDisparity+1)))<<endl;
	//cout <<"this new " <<hipMalloc(&L8,(sizeof(int)*(widthR)*(lengthR)*(maxDisparity+1)))<<endl;


	//Setting Up initial Info, this is done just once in the algorithm.
	initialInfo = (startInfo*)malloc((sizeof(startInfo))*pathNumber);
	getKernelInitialInformation(pathNumber,initialInfo,widthR,lengthR);
	hipMalloc(&initialInfoToKernel,(sizeof(startInfo))*pathNumber);
	hipMemcpy(initialInfoToKernel,initialInfo,(sizeof(startInfo))*pathNumber,hipMemcpyHostToDevice);

	KernelSemiGlobal<<<dimGrid2,dimBlock2>>>(costK,widthR,lengthR,initialInfoToKernel,maxDisparity,L1,L2,L3,L4/*,L5,L6,L7,L8*/);
	cout << "Synchronise status: "<<hipDeviceSynchronize() << endl;

	//------------Done-Second-Kernel-----------------------------
	//int* cost = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	//hipMemcpy(cost,costK,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1),hipMemcpyDeviceToHost);
	//Cost From GPU to host
	cout << "Done CUDA " << endl;
	int* L1S = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	int* L2S = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	int* L3S = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	int* L4S = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	//int* L5S = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	//int* L6S = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	//int* L7S = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	//int* L8S = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));

	hipMemcpy(L1S,L1,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1),hipMemcpyDeviceToHost);
	hipMemcpy(L2S,L2,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1),hipMemcpyDeviceToHost);
	hipMemcpy(L3S,L3,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1),hipMemcpyDeviceToHost);
	hipMemcpy(L4S,L4,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1),hipMemcpyDeviceToHost);
	//hipMemcpy(L5S,L5,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1),hipMemcpyDeviceToHost);
	//hipMemcpy(L6S,L6,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1),hipMemcpyDeviceToHost);
	//hipMemcpy(L7S,L7,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1),hipMemcpyDeviceToHost);
	//hipMemcpy(L8S,L8,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1),hipMemcpyDeviceToHost);
	cout << "Path Number " << pathNumber << endl;
	cout << L1S[widthR*(369+99*(lengthR))+100] << endl;
	//Aggregate Cost---------------------------------------------------------------------------------
	/*int* L1 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L1,left.cols-decreseX,left.rows-decreseY,maxDisparity,-1,-1);
	cout << "Done" << endl;

	int* L2 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L2,left.cols-decreseX,left.rows-decreseY,maxDisparity,0,-1);
	cout << "Done" << endl;

	int* L3 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L3,left.cols-decreseX,left.rows-decreseY,maxDisparity,1,-1);
	cout << "Done" << endl;

	int* L4 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L4,left.cols-decreseX,left.rows-decreseY,maxDisparity,-1,0);
	cout << "Done" << endl;

	int* L5 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L5,left.cols-decreseX,left.rows-decreseY,maxDisparity,1,0);
	cout << "Done" << endl;

	int* L6 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L6,left.cols-decreseX,left.rows-decreseY,maxDisparity,-1,1);
	cout << "Done" << endl;

	int* L7 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L7,left.cols-decreseX,left.rows-decreseY,maxDisparity,0,1);
	cout << "Done" << endl;

	int* L8 = (int*)malloc((sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1));
	AggregateCostCom(cost,L8,left.cols-decreseX,left.rows-decreseY,maxDisparity,1,1);
	cout << "Done" << endl;*/
	//Done Aggregate Cost----------------------------------------------------------------------------



	//Disparity Selection--------------------------------------------------------------------------------------------
	Mat disparity = DisparitySelectionP(L1S,L2S,L3S,L4S/*,L5S,L6S,L7S,L8S*/,maxDisparity,left.cols-decreseX,left.rows-decreseY);
	gettimeofday(&timstr, NULL);
	double end = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
	printf("Elapsed time Local Matching:\t\t\t%.6lf (s)\n", end - begin);
	//---------------------------------------------------------------------------------------------------------------

	namedWindow("SMG");
	imshow("SMG",disparity);
	//imwrite("disparity.png",disparity);
	waitKey(0);


	//free Memory
	hipFree(leftC);
	hipFree(rightC);
	hipFree(censusLa);
	hipFree(censusRa);
	hipFree(costK);
	//free(cost);
	free(L1S);
	free(L2S);
	free(L3S);
	free(L4S);
	/*free(L5S);
	free(L6S);
	free(L7S);
	free(L8S);*/

	hipFree(L1);
	hipFree(L2);
	hipFree(L3);
	hipFree(L4);
	/*hipFree(L5);
	hipFree(L6);
	hipFree(L7);
	hipFree(L8);*/

	hipFree(initialInfoToKernel);

	free(initialInfo);
}

__host__ pointCoo getPoint(int ID, int pathX, int pathY, int width, int length){
	pointCoo point;
	int startX;
	int startY;
	//Initial Point, this is the corner of the diagonals
	if(ID == 0){
		startX = 0;
		startY = 0;
		if(pathX == -1){
			startX = width - 1;
		}
		if(pathY == -1){
			startY = length - 1;
		}
	}
	//As diagonals go through the width and length, this needs to be divided into two segments
	//Segment 1 across the x axis of the image, fix y coordinate of the image
	else if (ID < width){
		startX = ID;
		startY = 0;
		if(pathX == -1){
			startX = startX - 1;
		}
		if(pathY == -1){
			startY = length-1;
		}
	}
	//Segment 2 across the y axis of the image, fix x coordinate
	else if (ID < width + length - 1){
		int newID = ID - width;
		startX = 0;
		startY = newID;
		if (pathX == -1){
			startX = width -1;
		}
		if (pathY == -1){
			startY = startY -1;
		}
	}

	point.x = startX;
	point.y = startY;
	return point;
}

__host__ void getKernelInitialInformation(int pathNumber, startInfo* pixelDi, int width, int length) {
	for (int blockID = 0; blockID < pathNumber; blockID++) {
		int ID = blockID;
		int LA = blockID; // Location in Array

		if (blockID < width) {
			pixelDi[LA].startX = ID;
			pixelDi[LA].startY = 0;
			pixelDi[LA].directionX = 0;
			pixelDi[LA].directionY = 1;
		} else if (blockID < 2 * width) {
			ID = blockID - width;
			pixelDi[LA].startX = ID;
			pixelDi[LA].startY = length - 1;
			pixelDi[LA].directionX = 0;
			pixelDi[LA].directionY = -1;
		} else if (blockID < 2 * width + length) {
			ID = blockID - 2 * width;
			pixelDi[LA].startX = 0;
			pixelDi[LA].startY = ID;
			pixelDi[LA].directionX = 1;
			pixelDi[LA].directionY = 0;
		} else if (blockID < 2 * width + 2 * length) {
			ID = blockID - 2 * width - length;
			pixelDi[LA].startX = width - 1;
			pixelDi[LA].startY = ID;
			pixelDi[LA].directionX = -1;
			pixelDi[LA].directionY = 0;
		} else if (blockID < 2 * width + 2 * length + (width + length - 1)) {
			cout << "Should not happen \n";
			ID = blockID - 2 * width - 2 * length;
			pixelDi[LA].directionX = 1;
			pixelDi[LA].directionY = 1;
			pointCoo point = getPoint(ID, 1, 1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else if (blockID < 2 * width + 2 * length + 2 * (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length - (width + length - 1);
			pixelDi[LA].directionX = -1;
			pixelDi[LA].directionY = -1;
			pointCoo point = getPoint(ID, -1, -1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else if (blockID < 2 * width + 2 * length + 3 * (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length - 2 * (width + length - 1);
			pixelDi[LA].directionX = 1;
			pixelDi[LA].directionY = -1;
			pointCoo point = getPoint(ID, 1, -1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else if (blockID < 2 * width + 2 * length + 4 * (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length - 3 * (width + length - 1);
			pixelDi[LA].directionX = -1;
			pixelDi[LA].directionY = 1;
			pointCoo point = getPoint(ID, -1, 1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else {
			printf("ifs are wrong \n");
		}
	}
}


//Disparity Selection Process
__host__ Mat DisparitySelectionP(int* L1, int* L2, int* L3, int* L4/*,int* L5, int* L6, int* L7, int* L8*/, int maxDisparity, int width, int length){
	Mat disparity(length,width,CV_8U);

	for(int y=0; y<length; y++){
		for(int x=0; x<width; x++){
			int costA = 99999;
			int disPix = 0;
			for (int d = 0; d<maxDisparity; d++){
				int sumAgg = L1[width*(y+d*length)+x] + L2[width*(y+d*(length))+x]+
						L3[width*(y+d*length)+x] + L4[width*(y+d*(length))+x] /*+ L5[width*(y+d*(length))+x]+
						L6[width*(y+d*length)+x] + L7[width*(y+d*(length))+x] + L8[width*(y+d*length)+x]*/;
				if(x == 100 && y == 369 && d == 99){
					cout << x << " " << y<< " : "<<L1[width*(y+d*length)+x] << endl;
					cout << width << " " << length << endl;
				}
				if (sumAgg < costA){
					costA = sumAgg;
					disPix = d;
				}
 			}
			//cout << "disparity pixel " <<x << " " << y << " is " << disPix << " with value "<< costA<< endl;
			disparity.at<uchar>(y,x) = disPix;
		}
	}
	return disparity;
}
//Gets the cost Computation Across all Paths
__host__ void AggregateCostCom(int* cost, int* L, int width, int length, int maxDisparity, int directionx, int directiony){
	//penalties
/*	int p1 = 10;
	int p2 = 100;*/

	int startX,startY, increaseX, increaseY;

	if(directionx <= 0){
		startX = 0;
		increaseX = 1;
	}
	else{
		startX = width-1;
		increaseX = -1;
	}
	if(directiony <= 0){
		startY = 0;
		increaseY = 1;
	}
	else{
		startY = length-1;
		increaseY = -1;
	}

	int* minimuns = (int*)malloc((sizeof(int))*(width)*(length));
	int x;
	int y = startY;

	for(int yC=0; yC<length; yC++){
		int influenceY = y + directiony;
		x = startX;
		for(int xC=0; xC<width; xC++){
			int influenceX = x + directionx;

			int minimunValue = 999999;

			if(influenceX > width || influenceX < 0 || influenceY > length || influenceY < 0){
				for (int d = 0; d<maxDisparity; d++){
					int costPixel = cost[width*(y+d*(length))+x];
					L[width*(y+d*(length))+x] = costPixel;
					if(costPixel < minimunValue){
						minimunValue = costPixel;
						//cout << "minimum ->  " << costPixel;
					}
				}
			}
			else{
				/*cout << x << " " << y <<endl;
				waitKey(0);*/
				for (int d = 0;  d<maxDisparity; d++){
					int costPixel = cost[width*(y+d*(length))+x];
					int currentD = L[width*(influenceY+d*(length))+influenceX];
					int previousD= (d-1>=0)? L[width*(influenceY+(d-1)*(length))+influenceX]: 8888;
					int nextD    = (d+1<maxDisparity)? L[width*(influenceY+(d+1)*(length))+influenceX] : 8888;

					//int minValue  = minArray(L,influenceX,influenceY,maxDisparity);
					int minValue =minimuns[influenceY*width+influenceX];
					int valueToAssign =  costPixel + minBetweenNumbersInt(currentD,nextD+p1,previousD+p1,minValue+p2) - minValue;

					L[width*(y+d*(length))+x] = valueToAssign;
					if(valueToAssign < minimunValue){
						minimunValue = valueToAssign;
					}

					/*if(valueToAssign <= 0){
						cout << x<<" "<<y<<"\t influence "<< influenceX << " " << influenceY << endl;
						cout << "Pixels: "<<x<<"-"<<y<<"\t"<<"Formula: " <<costPixel << "\t + min["<< currentD << "\t"<< previousD << "\t"
							<<nextD<<"\t"<<minValue<< "]" <<"\t Result Value: " <<valueToAssign<< "\t"<< d <<endl;
						waitKey(0);
					}*/
				}
				minimuns[y*width+x] = minimunValue;
			}
			x = x + increaseX;
		}
		y = y + increaseY;
	}


	cout << "done Aggregate Cost direction "<<directionx << " " << directiony << endl;
	free(minimuns);
}


__host__ int minBetweenNumbersInt(int a, int b, int c, int d){
	int min = a;

	if (min > b){
		min = b;
	}
	if (min > c){
		min = c;
	}
	if (min > d){
		min = d;
	}
	return min;
}


__host__ void DivideImagesCam(Mat completeImage, Mat* left, Mat* right){
	*left = completeImage(Rect(0,0,WIDTHIMAGE,LENGTHIMAGE));
	*right = completeImage(Rect(1280,0,WIDTHIMAGE,LENGTHIMAGE));
}

