#include <stdio.h>
#include <iostream>
#include <string.h>
#include <limits>

#include "opencv2/core/core.hpp"
#include <opencv2/imgproc/imgproc.hpp>
#include "opencv2/highgui/highgui.hpp"
#include <opencv2/contrib/contrib.hpp>
#include"string"

#include<sys/resource.h>
#include<time.h>
#include<sys/time.h>


#include "Comparison.h"
#include "Kernels.cuh"
#include "ZedCamera.h"
#include "Radar.h"

using namespace std;
using namespace cv;

#define threadx 16
#define thready 16

#define USECAMARA false
#define Camara 1
#define WIDTHIMAGE 1280
#define LENGTHIMAGE 720

__host__ void SGM();
__host__ Mat DisparitySelectionP(int* L1, int* L2, int* L3, int* L4,/*int* L5, int* L6, int* L7, int* L8,*/ int width, int length);
__host__ void DivideImagesCam(Mat completeImage, Mat* left, Mat* right);
__host__ Mat DisparitySelectionOneArray(int*L, int width, int length);

__host__ String numberOfZeros(int number);
__host__ String getImageLocation(int frame, String side);

__host__ pointCoo getPoint(int ID, int pathX, int pathY, int width, int length);
__host__ void getKernelInitialInformation(int pathNumber, startInfo* pixelDi, int width, int length);

//sl::Camera zed;

int main (int argc, char** argv){
	//Comparison Calculations
	//Comparison comp;
	//comp.CompareDisparities();
	//------------------------

	//Radar info
/*	Radar radar;
	int samples = 100;
	radar.startRadar();
	while(samples > 0){
		bool correctlyRead = radar.readInfo();
		while(correctlyRead == 0){
			bool correctlyRead = radar.readInfo();
		}
		waitKey(50);
		samples = samples - 1;
	}
	radar.closeRadar();*/


	SGM();
	return 0;
}

__host__ void SGM(){
	int frameNumber = 30;
	int frame = 0;

	struct timeval timstr;
	Mat left, right;
	Mat completeImage;
	//Texture Creation
	Mat leftD,rightD;
	uchar* imageLeftA,*imageRightA,*leftC,*rightC;


	//CensusDeclarations
	unsigned int* censusLa;
	unsigned int* censusRa;
	int* costK;


	//Maximum box value is depending on bytes used
	int BoxCostX = 9;
	int BoxCostY = 7;
	ZedCamera zed;
#if USECAMARA
	zed.initCamera();
	zed.grabImage();
	left = zed.getLeftImage();
	right = zed.getRightImage();
	/*char key = ' ';
	while(key != 'q'){
		zed.grabImage();
		left = zed.getLeftImage();
		right = zed.getRightImage();
		namedWindow("left");
		namedWindow("right");
		imshow("left", left);
		imshow("right", right);
		key = waitKey(10);
	}

	namedWindow("left");
	namedWindow("right");
	imshow("left", left);
	imshow("right", right);*/

#else
	//Reading Images
	left = imread("Images/KITTY/left/0000000000.png", CV_LOAD_IMAGE_COLOR);
	right= imread("Images/KITTY/right/0000000000.png", CV_LOAD_IMAGE_COLOR);
#endif

	//Initialisation Parameters---------------
	//First Kernel Params
	imageLeftA = (uchar*)malloc((sizeof(uchar))*(left.cols)*(left.rows));
	imageRightA= (uchar*)malloc((sizeof(uchar))*(right.cols)*(right.rows));

	int decreseX = BoxCostX/2 + BoxCostX/2;
	int decreseY = BoxCostY/2 + BoxCostY/2;

	int dimX = ((left.cols-decreseX) / threadx);
	int dimY = ((left.rows-decreseY) / thready);
	cout <<"Dimensions Grid: " << dimX << " " << dimY << endl;
	cout << "Dimensions Block: " << threadx << " " <<thready << endl;

	dim3 dimGrid(dimX,dimY);
	dim3 dimBlock(threadx,thready);
	//First Kernel, Census and cost Computation


	//Second Kernel Params
	int widthR = left.cols-decreseX;
	int lengthR = left.rows-decreseY;
	cout << "Disparity Size : " << widthR << " " << lengthR << endl;

	int threadNum = 258; // pathNumber is divisible by this, 437 blocks
	int pathNumber =/*(widthR+lengthR-1)*4 + */widthR *2 + lengthR*2;
	dim3 dimGrid2(pathNumber/threadNum);
	dim3 dimBlock2(threadNum);
	//Assigning Paths

	//Setting Up initial Info, this is done just once in the algorithm.


	int* L1S = (int*) malloc((sizeof(int)) * (left.cols - decreseX) * (left.rows - decreseY)* (maxDisparity + 1));
	//Done Initialisation Parameters----------

	//--------------------------------------------------------------------------------------------------
	//----------------------------Main Loop-------------------------------------------------------------
	//--------------------------------------------------------------------------------------------------
	while (frame < frameNumber || USECAMARA) {
		int* L1;
		cout <<"CUDA malloc " <<hipMalloc(&L1,(sizeof(int)*(widthR)*(lengthR)*(maxDisparity+1)))<<endl;
		cout << "CUDA malloc 1: "<<hipMalloc(&costK,(sizeof(int))*(left.cols-decreseX)*(left.rows-decreseY)*(maxDisparity+1)) << endl;
		hipMalloc(&censusLa,(sizeof(unsigned int))*(left.cols-decreseX)*(left.rows-decreseY));
		hipMalloc(&censusRa,(sizeof(unsigned int))*(right.cols-decreseX)*(right.rows-decreseY));


		startInfo* initialInfo, *initialInfoToKernel;
		initialInfo = (startInfo*) malloc((sizeof(startInfo)) * pathNumber);
		getKernelInitialInformation(pathNumber, initialInfo, widthR, lengthR);
		hipMalloc(&initialInfoToKernel, (sizeof(startInfo)) * pathNumber);
		hipMemcpy(initialInfoToKernel, initialInfo, (sizeof(startInfo)) * pathNumber, hipMemcpyHostToDevice);

#if USECAMARA
		zed.grabImage();
		left = zed.getLeftImage();
		right = zed.getRightImage();
#else
		left = imread(getImageLocation(frame,"left"), CV_LOAD_IMAGE_COLOR);
		right= imread(getImageLocation(frame,"right"), CV_LOAD_IMAGE_COLOR);
#endif
		//Timing
		gettimeofday(&timstr, NULL);
		double begin = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
		if (waitKey(1) >= 1) {
			cout << "Stoped at frame " << frame << endl;
			break;
		}

		namedWindow("left");
		namedWindow("right");
		imshow("left", left);
		imshow("right", right);
		cout << " here frame" << frame << endl;

		//Converting Images
#if USECAMARA
		Mat leftBlack;
		left.copyTo(leftBlack);
		Mat rightBlack;
		right.copyTo(rightBlack);
		namedWindow("leftB");
		namedWindow("rightB");
		//This line is wrong, not showing the correct image
		imshow("leftB", left);
		imshow("rightB", right);

		imageLeftA = left.data;
		imageRightA = right.data;
#else
		Mat leftBlack;
		cvtColor(left, leftBlack, CV_BGR2GRAY);
		Mat rightBlack;
		cvtColor(right, rightBlack, CV_BGR2GRAY);
		imageLeftA = leftBlack.data;
		imageRightA = rightBlack.data;
#endif
		hipMalloc(&leftC, (sizeof(uchar)) * (left.cols) * (left.rows));
		hipMalloc(&rightC, (sizeof(uchar)) * (left.cols) * (left.rows));
		hipMemcpy(leftC, imageLeftA, (sizeof(uchar)) * (left.cols) * (left.rows),hipMemcpyHostToDevice);
		hipMemcpy(rightC, imageRightA,(sizeof(uchar)) * (left.cols) * (left.rows),hipMemcpyHostToDevice);
		//Done COnverting Images

		gettimeofday(&timstr, NULL);
		double begin1 = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
		//CensusAndCostKernel<<<dimGrid,dimBlock>>>(BoxCostX,BoxCostY,widthR,lengthR,leftC,rightC,costK); //Not running
		KernelDisparityCalculations<<<dimGrid,dimBlock>>>(BoxCostX,BoxCostY,censusLa,censusRa,left.cols-decreseX,left.rows-decreseY,leftC,rightC,costK);
		cout << "Synchronise status kernel 1: " << hipDeviceSynchronize()<< "\n";
		//------------Done-First-Kernel-----------------------------
		gettimeofday(&timstr, NULL);
		double end1 = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
		printf("Elapsed time Census and Cost:\t\t\t%.6lf (s)\n", end1 - begin1);

		gettimeofday(&timstr, NULL);
		double begin2 = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
		//Second Kernel, Semi global matching and disparity Selection

		KernelSemiGlobal<<<dimGrid2,dimBlock2>>>(costK,widthR,lengthR,initialInfoToKernel,L1);
		cout << "Synchronise status kernel 2: " << hipDeviceSynchronize() << "\n";
		//------------Done-Second-Kernel-----------------------------

		gettimeofday(&timstr, NULL);
		double end2 = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
		printf("Elapsed time Aggregate Cost:\t\t\t%.6lf (s)\n", end2 - begin2);

		cout << "Done CUDA " << endl;

		hipMemcpy(L1S, L1,(sizeof(int)) * (left.cols - decreseX) * (left.rows - decreseY)* (maxDisparity + 1), hipMemcpyDeviceToHost);

		cout << "Path Number " << pathNumber << endl;

		//Disparity Selection--------------------------------------------------------------------------------------------
		Mat disparity = DisparitySelectionOneArray(L1S, widthR, lengthR);

		//---------------------------------------------------------------------------------------------------------------

		gettimeofday(&timstr, NULL);
		double end = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
		printf("Elapsed time Disparity:\t\t\t%.6lf (s)\n", end - begin);
		namedWindow("SMG");
		Mat display;
		double minVal, maxVal;
		minMaxIdx(disparity, &minVal, &maxVal);
		disparity.convertTo(display, CV_8UC3, 255 / (maxVal - minVal), -minVal);
		applyColorMap(display,display,COLORMAP_JET);
		imshow("SMG", display);
		//imwrite("disparit.png",disparity);
		frame ++;

		/*hipFree(leftC);
		hipFree(rightC);
		hipFree(censusLa);
		hipFree(censusRa);
		hipFree(costK);
		hipFree(L1);
		hipFree(initialInfoToKernel);*/
		free(initialInfo);
		hipDeviceReset();
	}
	//--------------------------------------------------------------------------------------------------
	//----------------------------Main Loop-------------------------------------------------------------
	//--------------------------------------------------------------------------------------------------

	//free Memory
	free(L1S);
	//free(imageLeftA);
	//free(imageRightA);


#if USECAMARA
	zed.closeCamera();
#endif
}


__host__ pointCoo getPoint(int ID, int pathX, int pathY, int width, int length){
	pointCoo point;
	int startX;
	int startY;
	//Initial Point, this is the corner of the diagonals
	if(ID == 0){
		startX = 0;
		startY = 0;
		if(pathX == -1){
			startX = width - 1;
		}
		if(pathY == -1){
			startY = length - 1;
		}
	}
	//As diagonals go through the width and length, this needs to be divided into two segments
	//Segment 1 across the x axis of the image, fix y coordinate of the image
	else if (ID < width){
		startX = ID;
		startY = 0;
		if(pathX == -1){
			startX = startX - 1;
		}
		if(pathY == -1){
			startY = length-1;
		}
	}
	//Segment 2 across the y axis of the image, fix x coordinate
	else if (ID < width + length - 1){
		int newID = ID - width;
		startX = 0;
		startY = newID;
		if (pathX == -1){
			startX = width -1;
		}
		if (pathY == -1){
			startY = startY -1;
		}
	}

	point.x = startX;
	point.y = startY;
	return point;
}

__host__ void getKernelInitialInformation(int pathNumber, startInfo* pixelDi, int width, int length) {
	for (int blockID = 0; blockID < pathNumber; blockID++) {
		int ID = blockID;
		int LA = blockID; // Location in Array

		if (blockID < width) {
			pixelDi[LA].startX = ID;
			pixelDi[LA].startY = 0;
			pixelDi[LA].directionX = 0;
			pixelDi[LA].directionY = 1;
		} else if (blockID < 2 * width) {
			ID = blockID - width;
			pixelDi[LA].startX = ID;
			pixelDi[LA].startY = length - 1;
			pixelDi[LA].directionX = 0;
			pixelDi[LA].directionY = -1;
		} else if (blockID < 2 * width + length) {
			ID = blockID - 2 * width;
			pixelDi[LA].startX = 0;
			pixelDi[LA].startY = ID;
			pixelDi[LA].directionX = 1;
			pixelDi[LA].directionY = 0;
		} else if (blockID < 2 * width + 2 * length) {
			ID = blockID - 2 * width - length;
			pixelDi[LA].startX = width - 1;
			pixelDi[LA].startY = ID;
			pixelDi[LA].directionX = -1;
			pixelDi[LA].directionY = 0;
		} else if (blockID < 2 * width + 2 * length + (width + length - 1)) {
			cout << "Should not happen \n";
			ID = blockID - 2 * width - 2 * length;
			pixelDi[LA].directionX = 1;
			pixelDi[LA].directionY = 1;
			pointCoo point = getPoint(ID, 1, 1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else if (blockID < 2 * width + 2 * length + 2 * (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length - (width + length - 1);
			pixelDi[LA].directionX = -1;
			pixelDi[LA].directionY = -1;
			pointCoo point = getPoint(ID, -1, -1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else if (blockID < 2 * width + 2 * length + 3 * (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length - 2 * (width + length - 1);
			pixelDi[LA].directionX = 1;
			pixelDi[LA].directionY = -1;
			pointCoo point = getPoint(ID, 1, -1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else if (blockID < 2 * width + 2 * length + 4 * (width + length - 1)) {
			ID = blockID - 2 * width - 2 * length - 3 * (width + length - 1);
			pixelDi[LA].directionX = -1;
			pixelDi[LA].directionY = 1;
			pointCoo point = getPoint(ID, -1, 1, width, length);
			pixelDi[LA].startX = point.x;
			pixelDi[LA].startY = point.y;
		} else {
			printf("ifs are wrong \n");
		}
	}
}


//Disparity Selection Process
__host__ Mat DisparitySelectionP(int* L1, int* L2, int* L3, int* L4/*,int* L5, int* L6, int* L7, int* L8*/, int width, int length){
	Mat disparity(length,width,CV_8U);

	for(int y=0; y<length; y++){
		for(int x=0; x<width; x++){
			int costA = 99999;
			int disPix = 0;
			for (int d = 0; d<maxDisparity; d++){
				int sumAgg = L1[width*(y+d*length)+x] + L2[width*(y+d*(length))+x]+
						L3[width*(y+d*length)+x] + L4[width*(y+d*(length))+x] /*+ L5[width*(y+d*(length))+x]+
						L6[width*(y+d*length)+x] + L7[width*(y+d*(length))+x] + L8[width*(y+d*length)+x]*/;
				if(x == 100 && y == 369 && d == 99){
					cout << x << " " << y<< " : "<<L1[width*(y+d*length)+x] << endl;
					cout << width << " " << length << endl;
				}
				if (sumAgg < costA){
					costA = sumAgg;
					disPix = d;
				}
 			}
			//cout << "disparity pixel " <<x << " " << y << " is " << disPix << " with value "<< costA<< endl;
			disparity.at<uchar>(y,x) = disPix;
		}
	}
	return disparity;
}

__host__ Mat DisparitySelectionOneArray(int*L, int width, int length){
	Mat disparity(length,width,CV_8U);
	for(int y=0;y<length;y++){
		for(int x=0;x<width;x++){
			int costA = 99999;
			int disPix = 0;
			for (int d=0;d<maxDisparity;d++){
				int value = L[width*(y+d*length)+x];
				if(value < costA){
					costA = value;
					disPix = d;
				}
			}
			disparity.at<uchar>(y,x) = disPix;
		}
	}
	return disparity;
}

__host__ void DivideImagesCam(Mat completeImage, Mat* left, Mat* right){
	*left = completeImage(Rect(0,0,WIDTHIMAGE,LENGTHIMAGE));
	*right = completeImage(Rect(1280,0,WIDTHIMAGE,LENGTHIMAGE));
}


//side = left, right
__host__ String getImageLocation(int frame, String side){
	//"Images/KITTY/left/0000000000.png"
	String imageLocation = "Images/KITTY/"+side+"/"+numberOfZeros(frame);
	ostringstream convert;
	convert << frame;
	imageLocation = imageLocation + convert.str() +  ".png";
	return imageLocation;
}

__host__ String numberOfZeros(int number){
	if(number < 10){
		return "000000000";
	}
	else if (number < 100){
		return "00000000";
	}
	else if (number < 1000){
		return "0000000";
	}
	else{
		return "000000";
	}
}


